#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/gather.h>

#define uint unsigned int

#define VERBOSE 0
#define BYTE_TO_BINARY_PATTERN "%c%c%c%c%c%c%c%c %c%c%c%c%c%c%c%c %c%c%c%c%c%c%c%c %c%c%c%c%c%c%c%c\t"
#define BYTE_TO_BINARY(byte)  \
  (byte & 0x80000000U ? '1' : '0'), \
  (byte & 0x40000000U ? '1' : '0'), \
  (byte & 0x20000000U ? '1' : '0'), \
  (byte & 0x10000000U ? '1' : '0'), \
  (byte & 0x8000000U ? '1' : '0'), \
  (byte & 0x4000000U ? '1' : '0'), \
  (byte & 0x2000000U ? '1' : '0'), \
  (byte & 0x1000000U ? '1' : '0') ,\
  (byte & 0x800000U ? '1' : '0'), \
  (byte & 0x400000U ? '1' : '0'), \
  (byte & 0x200000U ? '1' : '0'), \
  (byte & 0x100000U ? '1' : '0'), \
  (byte & 0x80000U ? '1' : '0'), \
  (byte & 0x40000U ? '1' : '0'), \
  (byte & 0x20000U ? '1' : '0'), \
  (byte & 0x10000U ? '1' : '0') ,\
  (byte & 0x8000U ? '1' : '0'), \
  (byte & 0x4000U ? '1' : '0'), \
  (byte & 0x2000U ? '1' : '0'), \
  (byte & 0x1000U ? '1' : '0'), \
  (byte & 0x800U ? '1' : '0'), \
  (byte & 0x400U ? '1' : '0'), \
  (byte & 0x200U ? '1' : '0'), \
  (byte & 0x100U ? '1' : '0'),\
  (byte & 0x80U ? '1' : '0'), \
  (byte & 0x40U ? '1' : '0'), \
  (byte & 0x20U ? '1' : '0'), \
  (byte & 0x10U ? '1' : '0'), \
  (byte & 0x8U ? '1' : '0'), \
  (byte & 0x4U ? '1' : '0'), \
  (byte & 0x2U ? '1' : '0'), \
  (byte & 0x1U ? '1' : '0') 



void parse_variables(const char* input_path, int* N, int* M, int* L) {
	FILE* file = fopen(input_path, "r");
	(void)fscanf(file, "%d,%d", N, L);
	if (*L < 1 || *N < 2) return;
	*M = (*L - 1) / 32 + 1;
	printf("vectors in file:\t%i\nbits in a vector:\t%i\nlength of a vector:\t%i\n\n", *N, *L, *M);
	fclose(file);
}
void read_data(thrust::host_vector<uint>& data, const char* input_path, int N, int M, int L) {
	FILE* file = fopen(input_path, "r");
	int c;
	do {
		c = fgetc(file);
	} while (c != '\n');
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			data[i + N * j] = 0u;
		}
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < L; j++) {
			c = fgetc(file);
			int index = i * M + (M * 32 - L + j) / 32;
			data[index] <<= 1;
			if (c == '1') {
				data[index]++;
			}
		}
		c = fgetc(file);
	}
	(void)fclose(file);
}


void print_data(thrust::device_vector<uint>& data, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			x = data[i * M + j];
			//printf("%08x ", x);
			printf(BYTE_TO_BINARY_PATTERN, BYTE_TO_BINARY(x));
		}
		printf("\n");
	}
}
void print_data(thrust::host_vector<uint>& data, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			x = data[i * M + j];
			//printf("%08x ", x);
			printf(BYTE_TO_BINARY_PATTERN, BYTE_TO_BINARY(x));
		}
		printf("\n");
	}
}
void print_data(thrust::device_vector<uint>& data, thrust::device_vector<uint>& permut, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			x = data[permut[i] * M + j];
			//printf("%08x ", x);
			printf(BYTE_TO_BINARY_PATTERN, BYTE_TO_BINARY(x));
		}
		printf("\n");
	}
}
void print_data(thrust::host_vector<uint>& data, thrust::host_vector<uint>& permut, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			x = data[permut[i] * M + j];
			//printf("%08x ", x);
			printf(BYTE_TO_BINARY_PATTERN, BYTE_TO_BINARY(x));
		}
		printf("\n");
	}
}
void print_data_32(thrust::device_vector<uint>& data, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			for (int k = 0; k < 32; k++) {
				x = data[i * M * 32 + j * 32 + k];
				printf("%x", x);
			}
			printf("\t");
		}
		printf("\n");
	}
}
void print_tuple(thrust::device_vector<uint>& data, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N * M * 32; i++) {
		for (int k = 0; k < 4; k++) {
			x = data[k * N * M * 32 + i];
			printf("% 3x", x);
		}
		printf("\n");
	}
}
void print_tuple(thrust::device_vector<uint>& data, thrust::device_vector<uint>& permut, int N, int M) {
	uint x;
	printf("\n------------\n");
	for (int i = 0; i < N * M * 32; i++) {
		for (int k = 0; k < 4; k++) {
			x = data[k * N * M * 32 + permut[i]];
			printf("% 3x", x);
		}
		printf("\n");
	}
}
struct extract_bit : public thrust::unary_function<uint, uint> {
	int bit = 0;
	__host__ __device__
		uint operator()(uint n)
	{
		return (n & (1U << bit)) >> bit;
	}
};
struct which_row : thrust::unary_function<int, int> {
	int row_length;

	__host__ __device__
		which_row(int row_length_) : row_length(row_length_) {}

	__host__ __device__
		int operator()(int idx) const {
		return idx / row_length;
	}
};
struct tuple_row4_iterator : thrust::unary_function<int, int> {
	int M;

	__host__ __device__
		tuple_row4_iterator(int M_) : M(M_) {}

	__host__ __device__
		int operator()(int idx) const {
		return M * 32 - 1 - (idx % (M * 32));
		//return idx;
	}
};
struct tuple_mult_iterator : thrust::unary_function<int, int> {
	int M;

	__host__ __device__
		tuple_mult_iterator(int M_) : M(M_) {}

	__host__ __device__
		int operator()(int idx) const {
		return idx / (M * 32);
		//return idx;
	}
};
struct iterator_modM32 : thrust::unary_function<int, int> {
	int M;

	__host__ __device__
		iterator_modM32(int M_) : M(M_) {}

	__host__ __device__
		int operator()(int idx) const {
		return idx % (32 * M);
		//return idx;
	}
};
struct reverse_bits : public thrust::unary_function<uint, uint> {
	__host__ __device__
		uint operator()(uint n)
	{
		uint rev = 0U;
		for (int i = 0; i < 32; i++) {
			rev <<= 1;
			if (n & 1U) {
				rev ^= 1U;
			}
			n >>= 1;
		}
		return rev;
	}
};
__host__
void scan_matrix_by_rows_logical_or(thrust::device_vector<uint>& u, int n, int m) {
	thrust::counting_iterator<int> c_first(0);
	thrust::transform_iterator<which_row, thrust::counting_iterator<int>> t_first(c_first, which_row(m));
	thrust::exclusive_scan_by_key(t_first, t_first + n * m, u.begin(), u.begin(), 0U, thrust::equal_to<int>(), thrust::logical_or<uint>());
}

int main() {
	int M, L, N;
	const char* input_path = "tests/test2.dat";
	parse_variables(input_path, &N, &M, &L);
	printf("\nLoading\n");

	thrust::device_vector<uint> d_giga_vector(M * N * 32 * 4);
	thrust::device_vector<uint> d_giga_index(M * N * 32);
	thrust::fill(d_giga_vector.begin(), d_giga_vector.end(), 0U);
	thrust::fill(d_giga_index.begin(), d_giga_index.end(), 0U);
	thrust::host_vector<uint> h_data(M * N);
	thrust::device_vector<uint> d_data_forw(M * N);
	thrust::device_vector<uint> d_xors_forw(M * N);
	thrust::device_vector<uint> d_permut_forw(N);
	thrust::device_vector<uint> d_data_back(M * N);
	thrust::device_vector<uint> d_xors_back(M * N);
	thrust::device_vector<uint> d_permut_back(N);
	thrust::device_vector<uint> d_temp1(N);
	thrust::device_vector<uint> d_temp2(N);
	thrust::device_vector<uint> d_index(N);

	read_data(h_data, input_path, N, M, L);

	thrust::copy(h_data.begin(), h_data.end(), d_data_forw.begin());
	thrust::transform(d_data_forw.begin(), d_data_forw.end(), d_data_back.begin(), reverse_bits());
	thrust::reverse(d_data_back.begin(), d_data_back.end());
	thrust::sequence(d_permut_forw.begin(), d_permut_forw.end());
	thrust::sequence(d_permut_back.rbegin(), d_permut_back.rend());
	if (VERBOSE)print_data(d_data_forw, d_permut_forw, N, M);
	if (VERBOSE)print_data(d_data_back, d_permut_back, N, M);
	printf("\nSorting\n");
	for (int i = M - 1; i >= 0; i--) {
		//printf("%d out of %d\n", i, M);
		thrust::sequence(d_index.begin(), d_index.end(), i, M);
		thrust::gather(d_index.begin(), d_index.end(), d_data_forw.begin(), d_temp1.begin());
		thrust::gather(d_permut_forw.begin(), d_permut_forw.end(), d_temp1.begin(), d_temp2.begin());
		thrust::sort_by_key(d_temp2.begin(), d_temp2.end(), d_permut_forw.begin(), thrust::less<uint>());

		thrust::sequence(d_index.begin(), d_index.end(), i, M);
		thrust::gather(d_index.begin(), d_index.end(), d_data_back.begin(), d_temp1.begin());
		thrust::gather(d_permut_back.begin(), d_permut_back.end(), d_temp1.begin(), d_temp2.begin());
		thrust::sort_by_key(d_temp2.begin(), d_temp2.end(), d_permut_back.begin(), thrust::less<uint>());
	}
	if (VERBOSE)print_data(d_data_forw, d_permut_forw, N, M);
	if (VERBOSE)print_data(d_data_back, d_permut_back, N, M);
	printf("\nXoring\n");
	for (int i = M - 1; i >= 0; i--) {
		//printf("%d out of %d\n", i, M);
		thrust::sequence(d_index.begin(), d_index.end(), i, M);
		thrust::gather(d_index.begin(), d_index.end(), d_data_forw.begin(), d_temp2.begin());
		thrust::gather(d_permut_forw.begin(), d_permut_forw.end(), d_temp2.begin(), d_temp1.begin());
		thrust::transform(d_temp1.begin(), d_temp1.end() - 1, d_temp1.begin() + 1, d_temp2.begin(), thrust::bit_xor<uint>());
		thrust::scatter(d_temp2.begin(), d_temp2.end(), d_index.begin(), d_xors_forw.begin());

		thrust::sequence(d_index.begin(), d_index.end(), i, M);
		thrust::gather(d_index.begin(), d_index.end(), d_data_back.begin(), d_temp2.begin());
		thrust::gather(d_permut_back.begin(), d_permut_back.end(), d_temp2.begin(), d_temp1.begin());
		thrust::transform(d_temp1.begin(), d_temp1.end() - 1, d_temp1.begin() + 1, d_temp2.begin(), thrust::bit_xor<uint>());
		thrust::scatter(d_temp2.begin(), d_temp2.end(), d_index.begin(), d_xors_back.begin());
	}
	if (VERBOSE)print_data(d_xors_forw, N, M);
	if (VERBOSE)print_data(d_xors_back, N, M);
	printf("\nPART 1\n");
	extract_bit op_bit;
	printf("\nReplicating\n");
	for (int i = 0; i < M; i++) {
		//printf("%d out of %d\n", i, M);
		thrust::sequence(d_index.begin(), d_index.end(), i, M);
		thrust::gather(d_index.begin(), d_index.end(), d_xors_forw.begin(), d_temp1.begin());
		for (int j = 0; j < 32; j++) {
			op_bit.bit = 32 - j - 1;
			thrust::sequence(d_index.begin(), d_index.end(), 32 * i + j, 32 * M);
			thrust::transform(d_temp1.begin(), d_temp1.end(), d_temp2.begin(), op_bit);
			thrust::scatter(d_temp2.begin(), d_temp2.end(), d_index.begin(), d_giga_vector.begin());
		}
	}
	printf("\nRowscan\n");
	if (VERBOSE)print_data_32(d_giga_vector, N, M);
	scan_matrix_by_rows_logical_or(d_giga_vector, N, M * 32);
	if (VERBOSE)print_data_32(d_giga_vector, N, M);
	printf("\nColscan\n");
	for (int i = 0; i < M * 32; i++) {
		thrust::sequence(d_index.begin(), d_index.end(), i, M * 32);
		thrust::gather(d_index.begin(), d_index.end(), d_giga_vector.begin(), d_temp1.begin());
		thrust::exclusive_scan(d_temp1.begin(), d_temp1.end(), d_temp1.begin(), 0U, thrust::plus<uint>());
		thrust::scatter(d_temp1.begin(), d_temp1.end(), d_index.begin(), d_giga_vector.begin());
	}
	if (VERBOSE)print_data_32(d_giga_vector, N, M);
	printf("\nGenerating Tuple\n");
	

	thrust::counting_iterator<int> reg(0);
	thrust::transform_iterator<tuple_mult_iterator, thrust::counting_iterator<int>> iter_mult(reg, tuple_mult_iterator(M));
	thrust::transform_iterator<iterator_modM32, thrust::counting_iterator<int>> iter_M32(reg, iterator_modM32(M));
	thrust::transform_iterator<tuple_row4_iterator, thrust::counting_iterator<int>> iter_row4(reg, tuple_row4_iterator(M));

	thrust::gather(iter_mult, iter_mult + N * M * 32, d_permut_forw.begin(), d_giga_index.begin());
	thrust::transform(d_giga_index.begin(), d_giga_index.end(), thrust::make_constant_iterator<uint>(M * 32), d_giga_index.begin(), thrust::multiplies<uint>());
	thrust::transform(d_giga_index.begin(), d_giga_index.end(), iter_M32, d_giga_index.begin(), thrust::plus<uint>());
	thrust::scatter(d_giga_vector.begin(), d_giga_vector.begin() + N * M * 32, d_giga_index.begin(), d_giga_vector.begin() + M * N * 32 * 2);

	printf("\nPART 2\n");
	printf("\nReplicating\n");
	for (int i = 0; i < M; i++) {
		thrust::sequence(d_index.begin(), d_index.end(), i, M);
		thrust::gather(d_index.begin(), d_index.end(), d_xors_back.begin(), d_temp1.begin());
		for (int j = 0; j < 32; j++) {
			op_bit.bit = 32 - j - 1;
			thrust::sequence(d_index.begin(), d_index.end(), 32 * i + j, 32 * M);
			thrust::transform(d_temp1.begin(), d_temp1.end(), d_temp2.begin(), op_bit);
			thrust::scatter(d_temp2.begin(), d_temp2.end(), d_index.begin(), d_giga_vector.begin());
		}
	}
	printf("\nRowscan\n");
	if (VERBOSE)print_data_32(d_giga_vector, N, M);
	scan_matrix_by_rows_logical_or(d_giga_vector, N, M * 32);
	if (VERBOSE)print_data_32(d_giga_vector, N, M);
	printf("\nColscan\n");

	for (int i = 0; i < M * 32; i++) {
		thrust::sequence(d_index.begin(), d_index.end(), i, M * 32);
		thrust::gather(d_index.begin(), d_index.end(), d_giga_vector.begin(), d_temp1.begin());
		thrust::exclusive_scan(d_temp1.begin(), d_temp1.end(), d_temp1.begin(), 0U, thrust::plus<uint>());
		thrust::scatter(d_temp1.begin(), d_temp1.end(), d_index.begin(), d_giga_vector.begin());
	}

	if (VERBOSE)print_data_32(d_giga_vector, N, M);
	printf("\nUpdating Tuple\n");


	thrust::gather(iter_mult, iter_mult + N * M * 32, d_permut_back.begin(), d_giga_index.begin());
	thrust::transform(d_giga_index.begin(), d_giga_index.end(), thrust::make_constant_iterator<uint>(M * 32), d_giga_index.begin(), thrust::multiplies<uint>());
	thrust::transform(d_giga_index.begin(), d_giga_index.end(), iter_row4, d_giga_index.begin(), thrust::plus<uint>());
	thrust::scatter(d_giga_vector.begin(), d_giga_vector.begin() + N * M * 32, d_giga_index.begin(), d_giga_vector.begin() + M * N * 32 * 3);

	thrust::copy(iter_mult, iter_mult + M * N * 32, d_giga_vector.begin() + M * N * 32 * 0);
	thrust::copy(iter_M32, iter_M32 + M * N * 32, d_giga_vector.begin() + M * N * 32 * 1);

	if (VERBOSE)print_tuple(d_giga_vector, N, M);
	printf("\nSorting Tuple\n");
	thrust::sequence(d_giga_index.begin(), d_giga_index.end());

	for (int i = 3; i > 0; i--) {
		thrust::gather(d_giga_index.begin(), d_giga_index.end(), d_giga_vector.begin() + N * M * 32 * i, d_giga_vector.begin());
		thrust::sort_by_key(d_giga_vector.begin(), d_giga_vector.begin() + N * M * 32, d_giga_index.begin(), thrust::less<uint>());
	}
	if (VERBOSE)print_tuple(d_giga_vector, d_giga_index, N, M);
	printf("\nPermuting Tuple\n");
	for (int i = 0; i < 4; i++) {
		thrust::gather(d_giga_index.begin(), d_giga_index.end(), d_giga_vector.begin() + N * M * 32 * i, d_giga_vector.begin());
		thrust::copy(d_giga_vector.begin(), d_giga_vector.begin() + N * M * 32, d_giga_vector.begin() + N * M * 32 * i);
	}
	printf("\nReducing Tuple\n");
	if (VERBOSE)print_tuple(d_giga_vector, N, M);
	for (int i = 3; i > 0; i--) {
		thrust::transform(d_giga_vector.begin() + N * M * 32 * i, d_giga_vector.begin() + N * M * 32 * (i + 1) - 1, d_giga_vector.begin() + N * M * 32 * i + 1, d_giga_index.begin() + 1, thrust::equal_to<uint>());
		thrust::copy(d_giga_index.begin() + 1, d_giga_index.end(), d_giga_vector.begin() + N * M * 32 * i + 1);
	}
	thrust::fill(d_giga_index.begin(), d_giga_index.end(), 1U);
	for (int i = 3; i > 0; i--) {
		thrust::transform(d_giga_vector.begin() + N * M * 32 * i, d_giga_vector.begin() + N * M * 32 * (i + 1), d_giga_index.begin(), d_giga_index.begin(), thrust::multiplies<uint>());
	}
	uint count = thrust::reduce(d_giga_index.begin() + 1, d_giga_index.end());
	if (VERBOSE)print_tuple(d_giga_vector, N, M);
	printf("\nPAIRS: %llu\n", count);
	getc(stdin);
	return 0;
}